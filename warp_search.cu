#include "hip/hip_runtime.h"
#include<vector>
#include"config.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include <memory>
#include"hipblas.h"

#include"smmh2.h"
#include"bin_heap.h"
#include"bloomfilter.h"
#include"blocked_bloomfilter.h"

#include<vector>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

#include "lightbam.cuh"
#include "gemm.cuh"

#define pq_idx_t u_int8_t
#define pq_value_t _Float32
#define value_t u_int8_t
#define idx_t u_int64_t
#define size_t int
#define pq_value_t _Float32
#define dist_t _Float32

#define FULL_MASK 0xffffffff
#define N_THREAD_IN_WARP 32
#define N_MULTIQUERY 1
#define CRITICAL_STEP (N_THREAD_IN_WARP/N_MULTIQUERY)
#define N_MULTIPROBE 1
#define FINISH_CNT 1

const int num_vertices = 1000;
const int dim = 128;
const int pq_dim = 128;
const int num_queries = 100;
const int degree = 100;
const int k = 256;
const int num_queues_per_ssd = 128;
const int queue_depth = 128;
const int max_io_size = 4096;

#define TOPK 100


template<size_t VALUE_SIZE, size_t INDEX_SIZE>
struct graph_node {
    value_t values[VALUE_SIZE];
    idx_t indexes[INDEX_SIZE];
};

template<class A,class B>
struct KernelPair{
    A first;
    B second;
	
	__device__
	KernelPair(){}


	__device__
    bool operator <(KernelPair& kp) const{
        return first < kp.first;
    }


	__device__
    bool operator >(KernelPair& kp) const{
        return first > kp.first;
    }
};

__device__ void computePQTable(
    pq_value_t* d_pq_centroid,  // PQ中心表,在GM中
    value_t* d_query,           // 单个查询向量
    pq_value_t* pq_table,       // PQ table的结果，已预分配内存，在shared_memory中
    int pq_dim,                      // 子空间数量
    int k,                      // 每个子空间的中心数量
    int dim,                    // 查询向量的维度
    int tid)                    
{
    int subvector_dim = dim / pq_dim; // 子空间的维度
    int step = 32;


	for(int i = 0;i < N_MULTIQUERY;++i){
		for (int subvector_idx = 0; subvector_idx < pq_dim; ++subvector_idx) {
			for (int centroid_idx = tid; centroid_idx < k; centroid_idx += step) {
				pq_value_t distance = 0;
				for (int dim_idx = 0; dim_idx < subvector_dim; ++dim_idx) {
					int query_dim_idx = i * dim + subvector_idx * subvector_dim + dim_idx;
					int centroid_dim_idx = (subvector_idx * k + centroid_idx) * subvector_dim + dim_idx;
					
					pq_value_t diff = d_query[query_dim_idx] - d_pq_centroid[centroid_dim_idx];
					distance += diff * diff;
				}
				
				pq_table[i * dim * k * pq_dim + subvector_idx * k + centroid_idx] = distance;
			}
		}
	}
}

__device__ static void read_data(uint64_t start_lb, uint64_t num_lb, IoQueuePair *ssdqp, uint64_t *prp1)
{
    uint32_t cid;
	// otherwise require cross-block synchronization
	assert(blockIdx.x < num_queues_per_ssd);
	assert(max_io_size <= AEOLUS_HOST_PGSIZE * 2);
    int global_queue_id = blockIdx.x;
    uint64_t global_pos = (uint64_t)global_queue_id * queue_depth;
	uint64_t offset = global_pos * max_io_size;
    uint64_t io_addr = prp1[offset / AEOLUS_DEVICE_PGSIZE] + offset % AEOLUS_DEVICE_PGSIZE;
	offset += AEOLUS_HOST_PGSIZE;
    uint64_t io_addr2 = prp1[offset / AEOLUS_DEVICE_PGSIZE] + offset % AEOLUS_DEVICE_PGSIZE;
    ssdqp[global_queue_id].submit(cid, NVME_OPCODE_READ, io_addr, io_addr2, start_lb & 0xffffffff, (start_lb >> 32) & 0xffffffff, NVME_RW_LIMITED_RETRY_MASK | (num_lb - 1));
    uint32_t status;
    ssdqp[global_queue_id].poll(status, cid);
    if (status != 0)
    {
        printf("read/write failed with status 0x%x\n", status);
        assert(0);
    }
}

__global__
void warp_independent_search_kernel(pq_idx_t* d_data,value_t* d_query,idx_t* d_result,pq_value_t* d_pq_centroid, int num_query, IoQueuePair *ssdqp, uint64_t *prp1, void *iobuf){
	const int QUEUE_SIZE = TOPK;
    int bid = blockIdx.x * N_MULTIQUERY;
	const int step = N_THREAD_IN_WARP;
    int tid = threadIdx.x;
	int cid = tid / CRITICAL_STEP;
	int subtid = tid % CRITICAL_STEP;
#define BLOOM_FILTER_BIT64 8
#define BLOOM_FILTER_BIT_SHIFT 3
#define BLOOM_FILTER_NUM_HASH 7

#ifndef __ENABLE_VISITED_DEL
#define HASH_TABLE_CAPACITY (TOPK*4*16)
#else
#define HASH_TABLE_CAPACITY (TOPK*4*2)
#endif

#ifdef __DISABLE_SELECT_INSERT
#undef HASH_TABLE_CAPACITY
#define HASH_TABLE_CAPACITY (TOPK*4*16+500)
#endif

    BlockedBloomFilter<BLOOM_FILTER_BIT64,BLOOM_FILTER_BIT_SHIFT,BLOOM_FILTER_NUM_HASH>* pbf;
    KernelPair<dist_t,idx_t>* q;
    KernelPair<dist_t,idx_t>* topk;
	value_t* dist_list;
	if(subtid == 0){
		dist_list = new value_t[FIXED_DEGREE * N_MULTIPROBE];
		q= new KernelPair<dist_t,idx_t>[QUEUE_SIZE + 2];
		topk = new KernelPair<dist_t,idx_t>[TOPK + 1];
    	pbf = new BlockedBloomFilter<BLOOM_FILTER_BIT64,BLOOM_FILTER_BIT_SHIFT,BLOOM_FILTER_NUM_HASH>();

	//pbf = new VanillaList();
	}
    __shared__ int heap_size[N_MULTIQUERY];
	int topk_heap_size;


	__shared__ int finished[N_MULTIQUERY];
	__shared__ idx_t index_list[N_MULTIQUERY][FIXED_DEGREE * N_MULTIPROBE];
	__shared__ char index_list_len[N_MULTIQUERY];
	// __shared__ pq_value_t pq_table[N_MULTIQUERY][pq_dim][k];
	extern __shared__ pq_value_t dynamic_shared_memory[];
	pq_value_t (*pq_table)[pq_dim][k] = (pq_value_t (*)[pq_dim][k])dynamic_shared_memory;
	
	value_t start_distance;
	__syncthreads();

	computePQTable(d_pq_centroid,d_query,&pq_table[0][0][0],pq_dim,k,dim,tid);
	__syncthreads();

	value_t tmp[N_MULTIQUERY];
	for(int j = 0;j < N_MULTIQUERY;++j){
		tmp[j] = 0;
		for(int i = tid;i < dim;i += step){
			tmp[j] += (pq_table[j][i][d_data[i]]) * (pq_table[j][i][d_data[i]]); 

		}
		for (int offset = 16; offset > 0; offset /= 2){
				tmp[j] += __shfl_xor_sync(FULL_MASK, tmp[j], offset);
		}
	}
	if(subtid == 0){
		start_distance = tmp[cid];
	}
	__syncthreads();
	
	if(subtid == 0){
    	heap_size[cid] = 1;
		topk_heap_size = 0;
		finished[cid] = false;
		dist_t d = start_distance;
		KernelPair<dist_t,idx_t> kp;
		kp.first = d;
		kp.second = 0;
		smmh2::insert(q,heap_size[cid],kp);
		pbf->add(0);
	}
	__syncthreads();
    while(heap_size[cid] > 1){
		index_list_len[cid] = 0;
		int current_heap_elements = heap_size[cid] - 1;
		for(int k = 0;k < N_MULTIPROBE && k < current_heap_elements;++k){
			KernelPair<dist_t,idx_t> now;
			if(subtid == 0){
				now = smmh2::pop_min(q,heap_size[cid]);
#ifdef __ENABLE_VISITED_DEL
				pbf->del(now.second);
#endif
				if(k == 0 && topk_heap_size == TOPK && (topk[0].first <= now.first)){
					++finished[cid];
				}
			}
			__syncthreads();
			if(finished[cid] >= FINISH_CNT)
				break;
			if(subtid == 0){
				topk[topk_heap_size++] = now;
				push_heap(topk,topk + topk_heap_size);
#ifdef __ENABLE_VISITED_DEL
				pbf->add(now.second);
#endif
				if(topk_heap_size > TOPK){
#ifdef __ENABLE_VISITED_DEL
					pbf->del(topk[0].second);
#endif
					pop_heap(topk,topk + topk_heap_size);
					--topk_heap_size;
				}
				int num_lbs = max_io_size / AEOLUS_LB_SIZE;
				read_data(now.second*num_lbs,num_lbs,ssdqp,prp1);
				graph_node<dim,degree> *now_node = (graph_node<dim,degree>*)(iobuf+1ll*blockIdx.x*queue_depth*max_io_size);
				for(int i = 0;i < degree;++i){
					auto idx = now_node->indexes[i];
					if(subtid == 0){
						if(pbf->test(idx)){
							continue;
						}
#ifdef __DISABLE_SELECT_INSERT
						pbf->add(idx);
#endif
						index_list[cid][index_list_len[cid]++] = idx;
					}
				}
			}
		}
		if(finished[cid] >= FINISH_CNT)
			break;
		__syncthreads();

		for(int nq = 0;nq < N_MULTIQUERY;++nq){
			for(int i = 0;i < index_list_len[nq];++i){
				value_t tmp = 0;
				for(int j = tid;j < dim;j += step){
					tmp += pq_table[nq][j][d_data[index_list[nq][i] * dim + j]]*pq_table[nq][j][d_data[index_list[nq][i] * dim + j]];
				}
				for (int offset = 16; offset > 0; offset /= 2){
					tmp += __shfl_xor_sync(FULL_MASK, tmp, offset);
				}
				if(tid == nq * CRITICAL_STEP){
					dist_list[i] = tmp;
				}
			}
		}

		__syncthreads();

		if(subtid == 0){
			for(int i = 0;i < index_list_len[cid];++i){
				dist_t d = dist_list[i];
				KernelPair<dist_t,idx_t> kp;
				kp.first = d;
				kp.second = index_list[cid][i];

				if(heap_size[cid] >= QUEUE_SIZE + 1 && q[2].first < kp.first){
					continue;
				}
#ifdef __ENABLE_MULTIPROBE_DOUBLE_CHECK
				if(pbf->test(kp.second))
					continue;
#endif
				smmh2::insert(q,heap_size[cid],kp);
#ifndef __DISABLE_SELECT_INSERT
				pbf->add(kp.second);
#endif
				if(heap_size[cid] >= QUEUE_SIZE + 2){
#ifdef __ENABLE_VISITED_DEL
					pbf->del(q[2].second);
#endif
					smmh2::pop_max(q,heap_size[cid]);
				}
			}
		}
		__syncthreads();
    }

	if(subtid == 0){
		for(int i = 0;i < TOPK;++i){
			auto now = pop_heap(topk,topk + topk_heap_size - i);
			d_result[(bid + cid) * TOPK + TOPK - 1 - i] = now.second;
		}
		delete[] q;
		delete[] topk;
    	delete pbf;
    	delete[] dist_list;
	}
}


static void astar_multi_start_search_batch(const std::vector<std::vector<std::pair<int,value_t>>>& queries,int k,\
	std::vector<std::vector<idx_t>>& results,pq_idx_t* h_data,graph_node<dim,degree>* h_graph,pq_value_t* pq_centroid,int num){
	pq_idx_t* d_data;
	value_t* d_query;
	idx_t* d_result;
	pq_value_t* d_pq_centroid;
	// graph_node<dim,degree>* d_graph;
	
	hipMalloc(&d_data,sizeof(pq_idx_t*) * num * dim);
	// hipMalloc(&d_graph,sizeof(graph_node<dim,degree>) * num);
	hipMalloc(&d_pq_centroid,sizeof(pq_value_t) * 256 * dim);
	hipMemcpy(d_data,h_data,sizeof(pq_idx_t*) * num * dim,hipMemcpyHostToDevice);
	// hipMemcpy(d_graph,h_graph,sizeof(graph_node<dim,degree>) * num,hipMemcpyHostToDevice);
	hipMemcpy(d_pq_centroid,pq_centroid,sizeof(pq_value_t) * 256 * dim,hipMemcpyHostToDevice);


	std::unique_ptr<value_t[]> h_query = std::unique_ptr<value_t[]>(new value_t[queries.size() * dim]);
	memset(h_query.get(),0,sizeof(value_t) * queries.size() * dim);
	for(int i = 0;i < queries.size();++i){
		for(auto p : queries[i]){
			*(h_query.get() + i * dim + p.first) = p.second;
		}
	}
	std::unique_ptr<idx_t[]> h_result = std::unique_ptr<idx_t[]>(new idx_t[queries.size() * TOPK]);

	hipMalloc(&d_query,sizeof(value_t) * queries.size() * dim);
	hipMalloc(&d_result,sizeof(idx_t) * queries.size() * TOPK);
	
	hipMemcpy(d_query,h_query.get(),sizeof(value_t) * queries.size() * dim,hipMemcpyHostToDevice);

	// init ssd controller
	std::vector<Device *> devices{new Device(0)};
	Controller *ctrl = new ControllerDecoupled(devices, num_queues_per_ssd, max_io_size, queue_depth, AEOLUS_DIST_STRIPE, AEOLUS_BUF_PINNED);
	PinnedBuffer *buf = new PinnedBuffer(devices[0], 1ll * num_queues_per_ssd * queue_depth * max_io_size, max_io_size);

	// warp_independent_search_kernel<<<queries.size()/N_MULTIQUERY,32>>>(d_data,d_query,d_result,d_graph,d_pq_centroid,queries.size());
	warp_independent_search_kernel<<<queries.size()/N_MULTIQUERY,32,sizeof(pq_value_t) * N_MULTIQUERY * pq_dim * 256>>>(d_data,d_query,d_result,d_pq_centroid,queries.size(), ctrl->get_io_queue_pair(), buf->get_d_prp_phys(), *buf);

	hipMemcpy(h_result.get(),d_result,sizeof(idx_t) * queries.size() * TOPK,hipMemcpyDeviceToHost);

	results.clear();
	for(int i = 0;i < queries.size();++i){
		std::vector<idx_t> v(TOPK);
		for(int j = 0;j < TOPK;++j)
			v[j] = h_result[i * TOPK + j];
		results.push_back(v);
	}
	hipFree(d_data);
	hipFree(d_query);
	hipFree(d_result);
	// hipFree(d_graph);
	hipFree(d_pq_centroid);
}


int main() {


    // 内存中保存pq量化数据
    pq_idx_t* h_data = new pq_idx_t[num_vertices * pq_dim];
    for (int i = 0; i < num_vertices * pq_dim; ++i) {
        h_data[i] = static_cast<pq_idx_t>(rand());
    }
    
    // vamana图结构，包含完整向量和索引
    graph_node<dim,degree>* h_graph = new graph_node<dim,degree>[num_vertices];
    for (int i = 0; i < num_vertices; ++i) {
        for (int j = 0; j < dim; ++j)
            h_graph[i].values[j] = static_cast<value_t>(rand()); // Random vertex index
        for (int j = 0; j < degree; ++j)
            h_graph[i].indexes[j] = static_cast<idx_t>(rand()) % num_vertices;
    }

    // 查询
    std::vector<std::vector<std::pair<int,value_t>>> queries(num_queries);
    for (int i = 0; i < num_queries; ++i) {
        // Each query has random values for different dimensions
        for (int j = 0; j < dim; ++j) {
            queries[i].push_back(std::make_pair(j, static_cast<value_t>(rand())));
        }
    }

    // pq中心表
    pq_value_t* pq_centroid=new pq_value_t[256 * dim];
        for (int i = 0; i < 256 * dim; ++i) {
            pq_centroid[i] = static_cast<pq_value_t>(rand());
        }

    // 结果保存
    std::vector<std::vector<idx_t>> results;

    // Call the function
    astar_multi_start_search_batch(queries, TOPK, results, h_data, h_graph, pq_centroid, num_vertices);

    // 结果输出
    std::cout << "Results:" << std::endl;
    for (int i = 0; i < results.size(); ++i) {
        std::cout << "Query " << i << ":" << std::endl;
        for (int j = 0; j < results[i].size(); ++j) {
            std::cout << results[i][j] << " ";
        }
        std::cout << std::endl;
    }


    return 0;
}
